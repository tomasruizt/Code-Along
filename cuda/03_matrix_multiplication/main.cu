#include "hip/hip_runtime.h"
#define ARMA_ALLOW_FAKE_GCC
#include <stdio.h>
#include <armadillo>
#include <cmath>

__host__ __device__ int idx(int i, int j, int rowSize) {
    return i * rowSize + j;
}

__host__ __device__ int idx_col(int i, int j, int colSize) {
    return i + j * colSize;
}

void print_matrix(int m, int n, double *A, bool rowMajor = true) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (rowMajor) {
                printf("%.6f ", A[idx(i, j, n)]);
            } else {
                printf("%.6f ", A[idx_col(i, j, m)]);
            }
        }
        printf("\n");
    }
}

__global__ void matrix_mult(double *A, double *B, double *C, int m, int n, int p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < m && j < p) {
        double sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[idx_col(i, k, n)] * B[idx_col(k, j, p)];
        }
        C[idx_col(i, j, p)] = sum;
    }
}

// Add this constant for tile size
#define TILE_SIZE 16

__global__ void matrix_mult_tiled(double *A, double *B, double *C, int m, int n, int p) {
    __shared__ double A_tile[TILE_SIZE][TILE_SIZE];
    __shared__ double B_tile[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    double sum = 0.0;
    
    // Loop over tiles
    for (int tile = 0; tile < (n + TILE_SIZE - 1) / TILE_SIZE; tile++) {
        // Load tiles into shared memory
        if (row < m && (tile * TILE_SIZE + tx) < n) {
            A_tile[ty][tx] = A[idx_col(row, tile * TILE_SIZE + tx, n)];
        } else {
            A_tile[ty][tx] = 0.0;
        }
        
        if ((tile * TILE_SIZE + ty) < n && col < p) {
            B_tile[ty][tx] = B[idx_col(tile * TILE_SIZE + ty, col, p)];
        } else {
            B_tile[ty][tx] = 0.0;
        }
        
        __syncthreads();
        
        // Compute partial sum for this tile
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += A_tile[ty][k] * B_tile[k][tx];
        }
        
        __syncthreads();
    }
    
    // Write result
    if (row < m && col < p) {
        C[idx_col(row, col, p)] = sum;
    }
}

void cpu_matrix_mult(double *A, double *B, double *C, int m, int n, int p) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < p; j++) {
            double sum = 0;
            for (int k = 0; k < n; k++) {
                sum += A[idx_col(i, k, n)] * B[idx_col(k, j, p)];
            }
            C[idx_col(i, j, p)] = sum;
        }
    }
}

bool compare_matrices(double *A, double *B, int m, int n, double tolerance) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (std::abs(A[idx(i, j, n)] - B[idx(i, j, n)]) > tolerance) {
                return false;
            }
        }
    }
    return true;
}

int main() {
    int m = 2;
    int n = 2;
    int p = 2;
    // Attention! Armadillo stores matrices in column-major order
    arma::mat A(m, n, arma::fill::randu);
    arma::mat B = {{1, 2}, {3, 4}}; //(n, p, arma::fill::randu);

    A.print("Matrix A:");
    printf("\n");

    B.print("Matrix B:");
    printf("\n");
    
    // Armadillo multiplication
    arma::mat C_arma = A * B;
    C_arma.print("Armadillo result C:");
    printf("\n");

    // CPU multiplication
    arma::mat C_cpu(m, p);
    cpu_matrix_mult(A.memptr(), B.memptr(), C_cpu.memptr(), m, n, p);
    printf("CPU result:\n");
    print_matrix(m, p, C_cpu.memptr(), false);
    printf("\n");

    // CUDA multiplication
    double *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, m * n * sizeof(double));
    hipMalloc((void**)&B_d, n * p * sizeof(double));
    hipMalloc((void**)&C_d, m * p * sizeof(double));

    hipMemcpy(A_d, A.memptr(), m * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B.memptr(), n * p * sizeof(double), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x, (p + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_mult<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, m, n, p);
    
    arma::mat C_cuda(m, p);
    hipMemcpy(C_cuda.memptr(), C_d, m * p * sizeof(double), hipMemcpyDeviceToHost);

    printf("CUDA result:\n");
    print_matrix(m, p, C_cuda.memptr(), false);
    printf("\n");

    // Tiled CUDA multiplication
    arma::mat C_cuda_tiled(m, p);
    matrix_mult_tiled<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, m, n, p);
    
    hipMemcpy(C_cuda_tiled.memptr(), C_d, m * p * sizeof(double), hipMemcpyDeviceToHost);

    printf("CUDA Tiled result:\n");
    print_matrix(m, p, C_cuda_tiled.memptr(), false);
    printf("\n");

    // Compare results
    double tolerance = 1e-6;
    bool cpu_match = compare_matrices(C_arma.memptr(), C_cpu.memptr(), m, p, tolerance);
    bool cuda_match = compare_matrices(C_arma.memptr(), C_cuda.memptr(), m, p, tolerance);
    bool cuda_tiled_match = compare_matrices(C_arma.memptr(), C_cuda_tiled.memptr(), m, p, tolerance);

    printf("CPU result matches Armadillo: %s\n", cpu_match ? "Yes" : "No");
    printf("CUDA result matches Armadillo: %s\n", cuda_match ? "Yes" : "No");
    printf("CUDA Tiled result matches Armadillo: %s\n", cuda_tiled_match ? "Yes" : "No");

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;
}