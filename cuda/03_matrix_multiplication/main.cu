#include "hip/hip_runtime.h"
#define ARMA_ALLOW_FAKE_GCC
#include <stdio.h>
#include <armadillo>
#include <cmath>
#include <chrono>

__host__ __device__ int idx(int i, int j, int rowSize) {
    return i * rowSize + j;
}

__host__ __device__ int idx_col(int i, int j, int colSize) {
    return i + j * colSize;
}

__global__ void matrix_mult(float *A, float *B, float *C, int m, int n, int p) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < m && j < p) {
        float sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[idx(i, k, n)] * B[idx(k, j, p)];
        }
        C[idx(i, j, p)] = sum;
    }
}

#define TILE_SIZE 16

__global__ void matrix_mult_tiled(float *A, float *B, float *C, int m, int n, int p) {
    __shared__ float A_tile[TILE_SIZE][TILE_SIZE];
    __shared__ float B_tile[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int tile = 0; tile < (n + TILE_SIZE - 1) / TILE_SIZE; tile++) {
        if (row < m && (tile * TILE_SIZE + tx) < n) {
            A_tile[ty][tx] = A[idx(row, tile * TILE_SIZE + tx, n)];
        } else {
            A_tile[ty][tx] = 0.0f;
        }
        
        if ((tile * TILE_SIZE + ty) < n && col < p) {
            B_tile[ty][tx] = B[idx(tile * TILE_SIZE + ty, col, p)];
        } else {
            B_tile[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += A_tile[ty][k] * B_tile[k][tx];
        }
        
        __syncthreads();
    }
    
    if (row < m && col < p) {
        C[idx(row, col, p)] = sum;
    }
}

void cpu_matrix_mult(float *A, float *B, float *C, int m, int n, int p) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < p; j++) {
            float sum = 0;
            for (int k = 0; k < n; k++) {
                sum += A[idx_col(i, k, n)] * B[idx_col(k, j, p)];
            }
            C[idx_col(i, j, p)] = sum;
        }
    }
}

bool compare_matrices(float *A, float *B, int m, int n, float tolerance) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (std::abs(A[idx(i, j, n)] - B[idx(i, j, n)]) > tolerance) {
                return false;
            }
        }
    }
    return true;
}

void transpose_and_copy(float* dest, const float* src, int rows, int cols) {
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            dest[idx(i, j, cols)] = src[idx_col(i, j, rows)];
        }
    }
}

int main() {
    const int NUM_RUNS = 5;
    int m = 1024;
    int n = 1024;
    int p = 1024;
    
    printf("Matrix dimensions: %dx%d * %dx%d = %dx%d\n", m, n, n, p, m, p);
    printf("Running each method %d times...\n\n", NUM_RUNS);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpu_milliseconds = 0;
    
    // Initialize matrices with single precision
    arma::fmat A(m, n, arma::fill::randu);
    arma::fmat B(n, p, arma::fill::randu);

    // Armadillo multiplication timing
    double arma_total = 0;
    arma::fmat C_arma(m, p);
    for (int run = 0; run < NUM_RUNS; run++) {
        auto cpu_start = std::chrono::high_resolution_clock::now();
        C_arma = A * B;
        auto cpu_end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(cpu_end - cpu_start);
        arma_total += duration.count() / 1000.0;
    }
    printf("Armadillo average time: %.3f ms\n\n", arma_total / NUM_RUNS);

    // CPU multiplication timing
    double cpu_total = 0;
    arma::fmat C_cpu(m, p);
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_matrix_mult(A.memptr(), B.memptr(), C_cpu.memptr(), m, n, p);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(cpu_end - cpu_start);
    cpu_total += duration.count() / 1000.0;
    printf("CPU average time: %.3f ms\n\n", cpu_total);

    // CUDA setup
    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, m * n * sizeof(float));
    hipMalloc((void**)&B_d, n * p * sizeof(float));
    hipMalloc((void**)&C_d, m * p * sizeof(float));

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (p + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Allocate host memory for row-major matrices
    float *A_row = new float[m * n];
    float *B_row = new float[n * p];
    float *C_row = new float[m * p];
    
    transpose_and_copy(A_row, A.memptr(), m, n);
    transpose_and_copy(B_row, B.memptr(), n, p);

    hipEventRecord(start);
    hipMemcpy(A_d, A_row, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_row, n * p * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_milliseconds, start, stop);
    float h2d_transfer_time = gpu_milliseconds;

    // Warmup run for GPU
    matrix_mult<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, m, n, p);
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error calling matrix_mult(): %s\n", hipGetErrorString(error));
        exit(1);
    }

    // Regular CUDA multiplication timing
    float cuda_total = 0;
    arma::fmat C_cuda(m, p);
    for (int run = 0; run < NUM_RUNS; run++) {
        hipEventRecord(start);
        matrix_mult<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, m, n, p);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpu_milliseconds, start, stop);
        cuda_total += gpu_milliseconds;
    }
    
    hipEventRecord(start);
    hipMemcpy(C_cuda.memptr(), C_d, m * p * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_milliseconds, start, stop);
    float d2h_transfer_time = gpu_milliseconds;

    // Warmup run for tiled GPU
    matrix_mult_tiled<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, m, n, p);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error calling matrix_mult_tiled(): %s\n", hipGetErrorString(error));
        exit(1);
    }

    // Tiled CUDA multiplication timing
    float cuda_tiled_total = 0;
    arma::fmat C_cuda_tiled(m, p);
    for (int run = 0; run < NUM_RUNS; run++) {
        hipEventRecord(start);
        matrix_mult_tiled<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, m, n, p);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpu_milliseconds, start, stop);
        cuda_tiled_total += gpu_milliseconds;
    }
    
    hipEventRecord(start);
    hipMemcpy(C_cuda_tiled.memptr(), C_d, m * p * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_milliseconds, start, stop);
    float d2h_transfer_time_tiled = gpu_milliseconds;

    printf("\nPerformance Results:\n");
    printf("----------------------------------------------------------------------\n");
    printf("| Method      | Computation | H2D Transfer | D2H Transfer | Total    |\n");
    printf("|-------------|-------------|--------------|--------------|----------|\n");
    printf("| CUDA        | %11.3f | %12.3f | %12.3f | %8.3f |\n",
           cuda_total / NUM_RUNS,
           h2d_transfer_time,
           d2h_transfer_time,
           (cuda_total / NUM_RUNS) + h2d_transfer_time + d2h_transfer_time);
    printf("| CUDA Tiled  | %11.3f | %12.3f | %12.3f | %8.3f |\n",
           cuda_tiled_total / NUM_RUNS,
           h2d_transfer_time,
           d2h_transfer_time_tiled,
           (cuda_tiled_total / NUM_RUNS) + h2d_transfer_time + d2h_transfer_time_tiled);
    printf("----------------------------------------------------------------------\n");
    printf("All times in milliseconds (ms)\n\n");

    printf("CPU Results:\n");
    printf("Armadillo: %.3f ms\n", arma_total / NUM_RUNS);
    printf("Basic CPU: %.3f ms\n\n", cpu_total);

    float tolerance = 1e-4f;  // Increased tolerance for float comparison
    bool cpu_match = compare_matrices(C_arma.memptr(), C_cpu.memptr(), m, p, tolerance);

    transpose_and_copy(C_row, C_cuda.memptr(), m, p);
    bool cuda_match = compare_matrices(C_arma.memptr(), C_row, m, p, tolerance);

    transpose_and_copy(C_row, C_cuda_tiled.memptr(), m, p);
    bool cuda_tiled_match = compare_matrices(C_arma.memptr(), C_row, m, p, tolerance);

    printf("Verification Results:\n");
    printf("CPU result matches Armadillo: %s\n", cpu_match ? "Yes" : "No");
    printf("CUDA result matches Armadillo: %s\n", cuda_match ? "Yes" : "No");
    printf("CUDA Tiled result matches Armadillo: %s\n", cuda_tiled_match ? "Yes" : "No");

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    delete[] A_row;
    delete[] B_row;
    delete[] C_row;

    return 0;
}
