#include <stdio.h>
#include "operations.h"

int main() {
    int n = 5;
    int a[] = {1, 2, 3, 4, 5};
    int b[] = {1, 2, 3, 4, 5};
    int c[n];
    mult(a, b, c, n);

    // Device pointers
    int *a_d, *b_d, *c_d;

    // Allocate memory on the device
    hipMalloc((void**)&a_d, n * sizeof(int));
    hipMalloc((void**)&b_d, n * sizeof(int));
    hipMalloc((void**)&c_d, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(a_d, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n * sizeof(int), hipMemcpyHostToDevice);

    printf("a = ");
    print_vector(a, n);

    printf("b = ");
    print_vector(b, n);

    printf("c = ");
    print_vector(c, n);
    printf("\n");

    // Free device memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return 0;
}