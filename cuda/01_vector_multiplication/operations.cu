
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void plus(int* a, int* b, int* c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}

void print_vector(int *a, int n) {
    printf("[ ");
    for (int i = 0; i < n; i++) {
        printf("%d ", a[i]);
    }
    printf("]");
    printf("\n");
}