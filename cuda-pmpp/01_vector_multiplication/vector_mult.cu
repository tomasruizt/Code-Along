#include <stdio.h>
#include "operations.h"

int main() {
    int n = 5;
    int a[] = {1, 2, 3, 4, 5};
    int b[] = {1, 2, 3, 4, 5};
    int c[n];

    // Device pointers
    int *a_d, *b_d, *c_d;

    // Allocate memory on the device
    hipMalloc((void**)&a_d, n * sizeof(int));
    hipMalloc((void**)&b_d, n * sizeof(int));
    hipMalloc((void**)&c_d, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(a_d, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n * sizeof(int), hipMemcpyHostToDevice);
    
    int gridSize = (n + 31) / 32;
    plus<<<gridSize, 32>>>(a_d, b_d, c_d, n);
    hipMemcpy(c, c_d, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("a = ");
    print_vector(a, n);

    printf("b = ");
    print_vector(b, n);

    printf("c = ");
    print_vector(c, n);

    // Free device memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return 0;
}