
#include <hip/hip_runtime.h>
#include <stdio.h>

struct Sum {
    __device__ float operator()(float a, float b) { 
        return a + b; 
    }

    static __device__ void atomic_reduce(float* addr, float val) { 
        atomicAdd(addr, val); 
    }
};

struct Max {
    __device__ float operator()(float a, float b) { 
        return max(a, b); 
    }

    static __device__ void atomic_reduce(float* addr, float val) {
        // Like in https://stackoverflow.com/a/17401122/5730291
        unsigned int* address_as_uint = (unsigned int*)addr;
        unsigned int old = *address_as_uint;
        unsigned int assumed;
        do {
            assumed = old;
            old = atomicCAS(address_as_uint, assumed,
                __float_as_int(max(__int_as_float(assumed), val)));
        } while (assumed != old);
    }
};

template<typename ReduceOp>
using ReduceKernelFn = void (*)(float*, int, float*);

template<typename ReduceOp>
__global__ void cudaNaiveKernel(float* vec, int len, float* result) {
    unsigned int tx = threadIdx.x;  // Apparently the unsigned int is required
    unsigned int i = 2 * tx + (2 * blockIdx.x * blockDim.x);
    ReduceOp reduce;
    
    // The particular stride creates uncoalesced memory access
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (tx % stride == 0 && i + stride < len)
            vec[i] = reduce(vec[i], vec[i + stride]);
        __syncthreads();
    }
    if (tx == 0)
        atomicAdd(result, vec[i]);
}

template<typename ReduceOp>
__global__ void cudaContinguousKernel(float* vec, int len, float* result) {
    unsigned int tx = threadIdx.x;
    unsigned int i = tx + (2 * blockIdx.x * blockDim.x);
    ReduceOp reduce;

    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
        if (tx < stride && i + stride < len) {
            vec[i] = reduce(vec[i], vec[i + stride]);
        }
        __syncthreads();
    }
    if (tx == 0)
        atomicAdd(result, vec[i]);
}


const int BLOCK_SIZE = 256;  // to fit 3 per SM

struct rtx3090 {
    int threads_per_sm = 1536;
    int blocks_per_sm = threads_per_sm / BLOCK_SIZE;
    int numSM = 82;
};

template<typename ReduceOp>
__global__ void cudaSharedMemKernel(float* vec, int len, float* result) {
    __shared__ float vec_s[BLOCK_SIZE];
    unsigned int tx = threadIdx.x;
    unsigned int i = tx + (2 * blockIdx.x * blockDim.x);
    ReduceOp reduce;

    // populate shared memory
    vec_s[tx] = reduce(vec[i], vec[i + blockDim.x]);
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        __syncthreads();
        if (tx < stride && i + stride < len) {
            vec_s[tx] = reduce(vec_s[tx], vec_s[tx + stride]);
        }
    }
    if (tx == 0)
        ReduceOp::atomic_reduce(result, vec_s[tx]);
}

template<typename ReduceOp>
float cudaReduce(float *vec, int len, ReduceKernelFn<ReduceOp> reduce)
{
    float *sum_h = new float[1];
    float *sum_d, *vec_d;
    hipMalloc((void **)&sum_d, sizeof(float));
    hipMalloc((void **)&vec_d, len * sizeof(float));
    hipMemcpy(vec_d, vec, len * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 numBlocks(ceil(len / (2.0 * threadsPerBlock.x)));
    // printf("numBlocks: %d\n", numBlocks.x);
    // printf("threadsPerBlock: %d\n", threadsPerBlock.x);
    reduce<<<numBlocks, threadsPerBlock>>>(vec_d, len, sum_d);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    hipMemcpy(sum_h, sum_d, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(sum_d);
    hipFree(vec_d);
    return sum_h[0];
}

float* ones(int n)
{
    float* vec = new float[n];
    for (int i = 0; i < n; i++)
    {
        vec[i] = 1.0f;
    }
    return vec;
}

float cpu_sum(float* vec, int len) {
    float sum = 0.0f;
    for (int i = 0; i < len; i++) {
        sum += vec[i];
    }
    return sum;
}

float cpu_max(float* vec, int len) {
    float max = -INFINITY;
    for (int i = 0; i < len; i++) {
        max = std::max(max, vec[i]);
    }
    return max;
}

float* randu(int n) {
    srand(0);
    float* vec = new float[n];
    for (int i = 0; i < n; i++) {
        vec[i] = rand() / (float)RAND_MAX;
    }
    return vec;
}

int main() {
    auto gpu = rtx3090();
    int nBlocks = gpu.blocks_per_sm * gpu.numSM;
    int nThreads = gpu.threads_per_sm * gpu.numSM;
    printf("nBlocks: %d\n", nBlocks);
    printf("nThreads: %d\n", nThreads);
    int n = nThreads * 2;
    
    printf("vec size (n): %d\n", n);
    float* vec = randu(n);
    float sum = cpu_sum(vec, n);
    printf("CPU sum: %.2f\n", sum);

    // Use the naive kernel
    float sum_h = cudaReduce<Sum>(vec, n, cudaNaiveKernel<Sum>);
    printf("CUDA naive sum: %.2f\n", sum_h);

    float sum_h2 = cudaReduce<Sum>(vec, n, cudaContinguousKernel<Sum>);
    printf("CUDA contiguous sum: %.2f\n", sum_h2);

    float sum_h3 = cudaReduce<Sum>(vec, n, cudaSharedMemKernel<Sum>);
    printf("CUDA sharedmem sum: %.2f\n", sum_h3);

    float max = cpu_max(vec, n);
    printf("CPU max: %.6f\n", max);

    float max_h = cudaReduce<Max>(vec, n, cudaSharedMemKernel<Max>);
    printf("CUDA sharedmem max: %.6f\n", max_h);

    return 0;
}