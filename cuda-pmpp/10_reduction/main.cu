
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define a type for kernel sum functions
typedef void (*SumFn)(float* vec_d, int len, float* sum_d);

__global__ void cudaNaiveSumKernel(float* vec, int len, float* result) {
    unsigned int tx = threadIdx.x;  // Apparently the unsigned int is required
    // The particular stride creates uncoalesced memory access
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (tx % stride == 0)
            vec[2 * tx] += vec[2 * tx + stride];
        __syncthreads();
    }
    if (tx == 0)
        result[0] = vec[0];
}

void cudaNaiveSum(float* vec_d, int len, float* sum_d) {
    dim3 threadsPerBlock(len / 2);
    dim3 numBlocks(1);
    cudaNaiveSumKernel<<<numBlocks, threadsPerBlock>>>(vec_d, len, sum_d);
}

__global__ void cudaContinguousSumKernel(float* vec, int len, float* result) {
    unsigned int i = threadIdx.x;
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
        if (i < stride) {
            vec[i] += vec[i + stride];
        }
        __syncthreads();
    }
    if (i == 0)
        result[0] = vec[0];
}

void cudaContinguousSum(float* vec_d, int len, float* sum_d) {
    dim3 threadsPerBlock(len / 2);
    dim3 numBlocks(1);
    cudaContinguousSumKernel<<<numBlocks, threadsPerBlock>>>(vec_d, len, sum_d);
}

float cudaSum(float *vec, int len, SumFn sum)
{
    float *sum_h = new float[1];
    float *sum_d, *vec_d;
    hipMalloc((void **)&sum_d, sizeof(float));
    hipMalloc((void **)&vec_d, len * sizeof(float));
    hipMemcpy(vec_d, vec, len * sizeof(float), hipMemcpyHostToDevice);
    
    sum(vec_d, len, sum_d);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    hipMemcpy(sum_h, sum_d, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(sum_d);
    hipFree(vec_d);
    return sum_h[0];
}

float* arange(int n)
{
    float* vec = new float[n];
    for (int i = 0; i < n; i++)
    {
        vec[i] = i + 1;
    }
    return vec;
}

float cpu_sum(float* vec, int len) {
    float sum = 0.0f;
    for (int i = 0; i < len; i++) {
        sum += vec[i];
    }
    return sum;
}

int main() {
    int n = 1024 * 2;  // cudaNaiveSum and cudaContiguous only work on a single block atm
    float* vec = arange(n);
    float sum = cpu_sum(vec, n);
    printf("CPU sum: %.2f\n", sum);

    // Use the naive kernel
    float sum_h = cudaSum(vec, n, cudaNaiveSum);
    printf("CUDA naive sum: %.2f\n", sum_h);

    float sum_h2 = cudaSum(vec, n, cudaContinguousSum);
    printf("CUDA contiguous sum: %.2f\n", sum_h2);
    
    return 0;
}