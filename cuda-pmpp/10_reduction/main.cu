#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main.h"

int main() {
    int n = 1024 * 2;
    float* vec = arange(n);
    float sum = cpu_sum(vec, n);
    printf("CPU sum: %.2f\n", sum);

    float* sum_h = cuda_sum(vec, n);
    printf("CUDA sum: %.2f\n", sum_h[0]);
    return 0;
}

float* cuda_sum(float *vec, int len)
{
    float *sum_h = new float[1];
    float *sum_d, *vec_d;
    hipMalloc((void **)&sum_d, sizeof(float));
    hipMalloc((void **)&vec_d, len * sizeof(float));
    hipMemcpy(vec_d, vec, len * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(len / 2);
    dim3 numBlocks(1);
    cudaNaiveSum<<<numBlocks, threadsPerBlock>>>(vec_d, len, sum_d);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    hipMemcpy(sum_h, sum_d, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(sum_d);
    hipFree(vec_d);
    return sum_h;
}

float* arange(int n)
{
    float* vec = new float[n];
    for (int i = 0; i < n; i++)
    {
        vec[i] = i + 1;
    }
    return vec;
}

float cpu_sum(float* vec, int len) {
    float sum = 0.0f;
    for (int i = 0; i < len; i++) {
        sum += vec[i];
    }
    return sum;
}

__global__ void cudaNaiveSum(float* vec, int len, float* result) {
    unsigned int tx = threadIdx.x;  // Apparently the unsigned int is required
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (tx % stride == 0)
            vec[2 * tx] += vec[2 * tx + stride];
    }
    if (tx == 0)
        result[0] = vec[0];
}
