/// WORK IN PROGRESS


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void naiveSoftmax(float* x, float* s, int n) {
    int i = threadIdx.x;
    s[i] = exp(x[i]);
}

int main() {
    float x[] = {1, 2, 3, 4, 5};
    int n = sizeof(x) / sizeof(x[0]);
    printf("n = %d\n", n);
    float s[n];
    float* d_x;
    float* d_s;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_s, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    int numBlocks = 1;
    int numThreads = n;
    naiveSoftmax<<<numBlocks, numThreads>>>(d_x, d_s, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    hipMemcpy(s, d_s, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_s);
    for (int i = 0; i < n; i++) {
        printf("%f ", s[i]);
    }
    printf("\n");
    return 0;
}
