#include <stdio.h>
#include <stdlib.h>
#include "matmul.cuh"

void init_random_matrix(float* matrix, int size) {
    for (int i = 0; i < size * size; i++) {
        matrix[i] = (float)rand() / RAND_MAX;
    }
}

int main() {
    const int n = 1024;  // Single dimension for square matrices
    
    printf("Matrix dimensions: %d\n", n);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpu_milliseconds = 0;
    
    // Allocate and initialize host matrices
    float *A = new float[n * n];
    float *B = new float[n * n];
    float *C = new float[n * n];
    
    init_random_matrix(A, n);
    init_random_matrix(B, n);

    // CUDA setup
    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, n * n * sizeof(float));
    hipMalloc((void**)&B_d, n * n * sizeof(float));
    hipMalloc((void**)&C_d, n * n * sizeof(float));

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Copy data to device
    hipMemcpy(A_d, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Run basic matrix multiplication
    printf("\nRunning basic matrix multiplication...\n");
    hipEventRecord(start);
    matrix_mult<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, n, n, n);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Basic matrix multiplication kernel failed: %s\n", hipGetErrorString(error));
        return 1;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_milliseconds, start, stop);
    printf("Basic matrix multiplication time: %.2f ms\n", gpu_milliseconds);

    // Run tiled matrix multiplication
    printf("\nRunning tiled matrix multiplication...\n");
    hipEventRecord(start);
    matrix_mult_tiled<<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, n, n, n);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Tiled matrix multiplication kernel failed: %s\n", hipGetErrorString(error));
        return 1;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_milliseconds, start, stop);
    printf("Tiled matrix multiplication time: %.2f ms\n", gpu_milliseconds);

    // Cleanup
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
} 